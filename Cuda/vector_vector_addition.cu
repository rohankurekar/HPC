
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
using namespace std;


__global__ void sum(int*  input1,int*  input2, int *Out,int size)              //kernel
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
	printf("\nThread id%d",threadIdx.x);
	if(i<size)
    Out[i]= input1[i] + input2[i];
    __syncthreads();
}

int main()
{
  int i,n;
  cout<<"Enter no. of elements";
  cin>>n;                                       //read number of elements in vector
  int a[n],b[n],c[n];
  cout<<"Enter elements of first vector";
  for(int i=0;i<n;i++)
  cin>>a[i];                                    //read first vector
  
  cout<<"Enter elements of second vector";
  for(int i=0;i<n;i++)
  cin>>b[i];                                    //read second vector
  
  
  int *a1, *b1, *c1;                            
  hipMalloc((void **) &a1, n*sizeof(int));     //allocating memory
  hipMalloc((void **) &b1, n*sizeof(int));
  hipMalloc((void **) &c1, n*sizeof(int));
  
 
  
  
  for( i = 0 ; i < n ; i++)
  {
    
	c[i]=0;
  }
  
  
  hipMemcpy(a1 , a, n*sizeof(int),hipMemcpyHostToDevice);    //data transfer from host memory to device memory
  hipMemcpy(b1 , b, n*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(c1 , c, n*sizeof(int),hipMemcpyHostToDevice);
  
  sum<<<1,n>>>(a1,b1,c1,n);                                    //calling kernel
  hipDeviceSynchronize();
  hipMemcpy(c, c1, n*sizeof(int),hipMemcpyDeviceToHost);    //data transfer from device memory to host memory
  
  for( i = 0 ; i < n ; i++)
  {
    
	cout<<"\n "<<a[i]<<" + "<<b[i]<<" = "<<c[i];
  }
  
  hipFree(a1);         //freeing memory
  hipFree(b1);
  hipFree(c1);
 
  return 0;
}
